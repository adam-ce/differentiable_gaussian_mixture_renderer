#include "hip/hip_runtime.h"
/*****************************************************************************
 * Differentiable Gaussian Mixture Renderer
 * Copyright (C) 2023 Adam Celarek
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *****************************************************************************/

#include <stroke/unittest/gradcheck.h>

#include <catch2/catch_test_macros.hpp>
#include <dgmr/grad/utils.h>
#include <dgmr/utils.h>
#include <glm/glm.hpp>
#include <stroke/linalg.h>
#include <whack/random/generators.h>

using namespace dgmr::utils;

namespace {
template <glm::length_t n_dims, typename scalar_t, typename Generator>
glm::mat<n_dims, n_dims, scalar_t> host_random_matrix(Generator* rnd)
{
    glm::mat<n_dims, n_dims, scalar_t> mat;
    for (auto c = 0; c < n_dims; ++c) {
        for (auto r = 0; r < n_dims; ++r) {
            mat[c][r] = rnd->normal();
        }
    }
    return mat;
}

template <glm::length_t n_dims, typename scalar_t, typename Generator>
stroke::Cov<n_dims, scalar_t> host_random_cov(Generator* rnd)
{
    const auto mat = host_random_matrix<n_dims, scalar_t>(rnd);
    return stroke::Cov<n_dims, scalar_t>(mat * transpose(mat)) + stroke::Cov<n_dims, scalar_t>(0.05);
}

template <typename scalar_t, typename Generator>
Camera<scalar_t> random_camera(Generator* rnd)
{
    const scalar_t fovy = scalar_t(3.14 / 4);

    Camera<scalar_t> c;
    c.fb_height = 600;
    c.fb_width = 800;
    const auto aspect = scalar_t(c.fb_width / c.fb_height);
    c.tan_fovy = std::tan(fovy);
    c.tan_fovx = std::atan(c.tan_fovy) * aspect; // via https://stackoverflow.com/questions/5504635/computing-fovx-opengl
    c.focal_x = c.fb_width / (2.0f * c.tan_fovx);
    c.focal_y = c.fb_height / (2.0f * c.tan_fovy);
    c.view_matrix = glm::lookAt(rnd->normal3() * scalar_t(5.), rnd->normal3() * scalar_t(2.5), glm::normalize(rnd->normal3()));
    c.view_projection_matrix = glm::perspective(scalar_t(fovy), aspect, scalar_t(0.1), scalar_t(100.)) * c.view_matrix;
    return c;
}

void check_splat()
{
    using scalar_t = float;
    using vec3_t = glm::vec<3, scalar_t>;
    using cov3_t = stroke::Cov3<scalar_t>;

    whack::random::HostGenerator<scalar_t> rnd;

    for (int i = 0; i < 10; ++i) {
        const auto cam = random_camera<scalar_t>(&rnd);
        const auto fun = [cam](const whack::Tensor<scalar_t, 1>& input) {
            const auto [weight, pos, cov] = stroke::extract<scalar_t, vec3_t, cov3_t>(input);
            Gaussian2d<scalar_t> g = splat<scalar_t>(weight, pos, cov, cam);
            return stroke::pack_tensor<scalar_t>(g);
        };

        const auto fun_grad = [cam](const whack::Tensor<scalar_t, 1>& input, const whack::Tensor<scalar_t, 1>& grad_output) {
            const auto [weight, pos, cov] = stroke::extract<scalar_t, vec3_t, cov3_t>(input);
            const Gaussian2d<scalar_t> grad_incoming = stroke::extract<Gaussian2d<scalar_t>>(grad_output);
            const auto grad_outgoing = grad::splat<scalar_t>(weight, pos, cov, grad_incoming, cam);
            return stroke::pack_tensor<scalar_t>(grad_outgoing);
        };

        const auto test_data = stroke::pack_tensor<scalar_t>(
            rnd.normal(),
            rnd.normal3(),
            host_random_cov<3, scalar_t>(&rnd));
        stroke::check_gradient(fun, fun_grad, test_data, scalar_t(0.0000001));
    }
}

void check_project()
{
    using scalar_t = double;
    using vec3_t = glm::vec<3, scalar_t>;
    using cov3_t = stroke::Cov3<scalar_t>;

    whack::random::HostGenerator<scalar_t> rnd;

    for (int i = 0; i < 10; ++i) {
        const auto cam = random_camera<scalar_t>(&rnd);
        const auto fun = [cam](const whack::Tensor<scalar_t, 1>& input) {
            const auto pos = stroke::extract<vec3_t>(input);
            const auto p = dgmr::utils::project<scalar_t>(pos, cam.view_projection_matrix);
            return stroke::pack_tensor<scalar_t>(p);
        };

        const auto fun_grad = [cam](const whack::Tensor<scalar_t, 1>& input, const whack::Tensor<scalar_t, 1>& grad_output) {
            const auto pos = stroke::extract<vec3_t>(input);
            const auto grad_incoming = stroke::extract<glm::vec<3, scalar_t>>(grad_output);
            const auto grad_outgoing = grad::project<scalar_t>(pos, cam.view_projection_matrix, grad_incoming);
            return stroke::pack_tensor<scalar_t>(grad_outgoing);
        };

        const auto test_data = stroke::pack_tensor<scalar_t>(rnd.normal3());
        stroke::check_gradient(fun, fun_grad, test_data, scalar_t(0.0000001));
    }
}

} // namespace

TEST_CASE("dgmr splat gradient")
{
    check_splat();
}

TEST_CASE("dgmr project gradient")
{
    check_project();
}
