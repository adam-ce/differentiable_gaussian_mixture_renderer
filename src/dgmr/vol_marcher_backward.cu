#include "hip/hip_runtime.h"
/*****************************************************************************
 * Differentiable Gaussian Mixture Renderer
 * Copyright (C) 2023 Adam Celarek
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *****************************************************************************/

#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>

#include "constants.h"
#include "marching_steps.h"
#include "math.h"
#include "vol_marcher_backward.h"

#include <stroke/gaussian.h>
#include <stroke/linalg.h>
#include <whack/Tensor.h>
#include <whack/kernel.h>
#include <whack/torch_interop.h>

namespace {
using namespace dgmr;
using namespace dgmr::vol_marcher;
namespace gaussian = stroke::gaussian;

// my own:
STROKE_DEVICES glm::vec3 clamp_cov_scales(const glm::vec3& cov_scales)
{
    const auto max_value = stroke::min(50.f, glm::compMax(cov_scales));
    const auto min_value = max_value * 0.01f;
    return glm::clamp(cov_scales, min_value, max_value);
}
} // namespace

dgmr::vol_marcher::Gradients dgmr::vol_marcher::backward(const dgmr::vol_marcher::ForwardData& data, const dgmr::vol_marcher::ForwardCache& cache)
{
    const auto fb_width = data.framebuffer.size<2>();
    const auto fb_height = data.framebuffer.size<1>();
    const auto n_gaussians = data.gm_weights.size<0>();
    const float focal_y = fb_height / (2.0f * data.tan_fovy);
    const float focal_x = fb_width / (2.0f * data.tan_fovx);
    const auto aa_distance_multiplier = (config::filter_kernel_SD * data.tan_fovx * 2) / fb_width;

    constexpr dim3 render_block_dim = { render_block_width, render_block_height };
    constexpr auto render_block_size = render_block_width * render_block_height;
    constexpr auto render_n_warps = render_block_size / 32;
    static_assert(render_n_warps * 32 == render_block_size);
    const dim3 render_grid_dim = whack::grid_dim_from_total_size({ data.framebuffer.size<2>(), data.framebuffer.size<1>() }, render_block_dim);

    // geometry buffers, filled by the forward preprocess pass
    auto g_rects = whack::make_tensor_view<const glm::uvec2>(cache.rects_data, n_gaussians);
    auto g_rgb = whack::make_tensor_view<const glm::vec3>(cache.rgb_data, n_gaussians);
    auto g_rgb_sh_clamped = whack::make_tensor_view<const glm::vec<3, bool>>(cache.rgb_sh_clamped_data, n_gaussians);
    auto g_depths = whack::make_tensor_view<const float>(cache.depths_data, n_gaussians);
    auto g_points_xy_image = whack::make_tensor_view<const glm::vec2>(cache.points_xy_image_data, n_gaussians);
    auto g_inverse_filtered_cov3d = whack::make_tensor_view<const stroke::Cov3_f>(cache.inverse_filtered_cov3d_data, n_gaussians);
    auto g_filtered_masses = whack::make_tensor_view<const float>(cache.filtered_masses_data, n_gaussians);
    auto g_tiles_touched = whack::make_tensor_view<const uint32_t>(cache.tiles_touched_data, n_gaussians);
    auto g_point_offsets = whack::make_tensor_view<const uint32_t>(cache.point_offsets_data, n_gaussians);

    // // render backward
    // // Let each tile blend its range of Gaussians independently in parallel
    // {
    //     // Main rasterization method. Collaboratively works on one tile per
    //     // block, each thread treats one pixel. Alternates between fetching
    //     // and rasterizing data.

    //     const auto inversed_projectin_matrix = glm::inverse(data.proj_matrix);

    //     auto i_ranges = whack::make_tensor_view(i_ranges_data.device_vector(), render_grid_dim.y, render_grid_dim.x);
    //     whack::start_parallel(
    //         whack::Location::Device, render_grid_dim, render_block_dim, WHACK_DEVICE_KERNEL(=) {
    //             WHACK_UNUSED(whack_gridDim);
    //             // Identify current tile and associated min/max pixel range.
    //             const glm::uvec2 pix_min = { whack_blockIdx.x * whack_blockDim.x, whack_blockIdx.y * whack_blockDim.y };
    //             const glm::uvec2 pix_max = min(pix_min + glm::uvec2(whack_blockDim.x, whack_blockDim.y), glm::uvec2(fb_width, fb_height));
    //             const glm::uvec2 pix = pix_min + glm::uvec2(whack_threadIdx.x, whack_threadIdx.y);
    //             const glm::vec2 pix_ndc = glm::vec2(pix * glm::uvec2(2)) / glm::vec2(fb_width, fb_height) - glm::vec2(1);
    //             auto view_at_world = inversed_projectin_matrix * glm::vec4(pix_ndc, -1, 1.0);
    //             view_at_world /= view_at_world.w;

    //             const auto ray = stroke::Ray<3, float> { data.cam_poition, glm::normalize(glm::vec3(view_at_world) - data.cam_poition) };
    //             const unsigned thread_rank = whack_blockDim.x * whack_threadIdx.y + whack_threadIdx.x;

    //                    // Check if this thread is associated with a valid pixel or outside.
    //             bool inside = pix.x < fb_width && pix.y < fb_height;
    //             // Done threads can help with fetching, but don't rasterize
    //             bool done = !inside;

    //             const auto render_g_range = i_ranges(whack_blockIdx.y, whack_blockIdx.x);
    //             const auto n_rounds = ((render_g_range.y - render_g_range.x + render_block_size - 1) / render_block_size);
    //             auto n_toDo = render_g_range.y - render_g_range.x;

    //                    // Allocate storage for batches of collectively fetched data.
    //             __shared__ int collected_id[render_block_size];
    //             __shared__ float collected_3d_masses[render_block_size];
    //             __shared__ glm::vec3 collected_centroid[render_block_size];
    //             __shared__ stroke::Cov3<float> collected_inv_cov3[render_block_size];

    //             bool large_stepping_ongoing = true;
    //             float current_large_step_start = 0.f;
    //             // float accumulated_mass = 0;
    //             static constexpr auto mass_threshold = -gcem::log(config::transmission_threshold);

    //             glm::vec3 current_colour = glm::vec3(0);
    //             float current_transparency = 1;
    //             float current_mass = 0;

    //             while (large_stepping_ongoing) {
    //                 // Iterate over all gaussians and take the first config::n_large_steps larger than current_large_step_start
    //                 marching_steps::DensityArray<config::n_large_steps> sample_sections(current_large_step_start);
    //                 n_toDo = render_g_range.y - render_g_range.x;
    //                 bool done_1 = !inside;
    //                 for (unsigned i = 0; i < n_rounds; i++, n_toDo -= render_block_size) {
    //                     // End if entire block votes that it is done rasterizing
    //                     const int num_done = __syncthreads_count(done || done_1);
    //                     if (num_done == render_block_size)
    //                         break;

    //                            // Collectively fetch per-Gaussian data from global to shared
    //                     const int progress = i * render_block_size + thread_rank;
    //                     if (render_g_range.x + progress < render_g_range.y) {
    //                         unsigned coll_id = b_point_list(render_g_range.x + progress);
    //                         assert(coll_id < n_gaussians);
    //                         collected_id[thread_rank] = coll_id;
    //                         collected_centroid[thread_rank] = data.gm_centroids(coll_id);
    //                         collected_inv_cov3[thread_rank] = g_inverse_filtered_cov3d(coll_id);
    //                         collected_3d_masses[thread_rank] = g_filtered_masses(coll_id);
    //                     }
    //                     __syncthreads();

    //                     if (done || done_1)
    //                         continue;

    //                            // Iterate over current batch
    //                     for (unsigned j = 0; j < min(render_block_size, n_toDo); j++) {
    //                         const auto gaussian1d = gaussian::intersect_with_ray_inv_C(collected_centroid[j], collected_inv_cov3[j], ray);
    //                         const auto sd = stroke::sqrt(gaussian1d.C);

    //                                // if (sample_sections.size() == config::n_large_steps && g_depths(collected_id[j]) > sample_sections[config::n_large_steps - 1]) {
    //                                //     done_1 = true;
    //                                //     break;
    //                                // }
    //                         if (sample_sections.end() < g_depths(collected_id[j])) {
    //                             done_1 = true;
    //                             break;
    //                         }

    //                         auto mass_on_ray = gaussian1d.weight * collected_3d_masses[j];
    //                         if (mass_on_ray <= 1.1f / 255.f || mass_on_ray > 1'000)
    //                             continue;
    //                         if (gaussian1d.C + vol_marcher::config::workaround_variance_add_along_ray <= 0)
    //                             continue;
    //                         if (stroke::isnan(gaussian1d.centre))
    //                             continue;

    //                         const float start = gaussian1d.centre - sd * config::gaussian_relevance_sigma;
    //                         const float end = gaussian1d.centre + sd * config::gaussian_relevance_sigma;
    //                         const float delta_t = (sd * config::gaussian_relevance_sigma * 2) / (config::n_steps_per_gaussian - 1);

    //                         sample_sections.put({ start, end, delta_t });
    //                     }
    //                 }

    //                        // iterate again, and compute linear interpolations
    //                 const auto bin_borders = marching_steps::sample<config::n_small_steps>(sample_sections);
    //                 whack::Array<glm::vec4, config::n_small_steps> bin_eval = {};

    //                 float dbg_mass_in_bins_closeed = 0;
    //                 float dbg_mass_in_bins_numerik_1 = 0;

    //                        // Iterate over batches until all done or range is complete: rasterise into bins
    //                 n_toDo = render_g_range.y - render_g_range.x;
    //                 for (unsigned i = 0; i < n_rounds; i++, n_toDo -= render_block_size) {
    //                     // End if entire block votes that it is done rasterizing
    //                     const int num_done = __syncthreads_count(done);
    //                     if (num_done == render_block_size)
    //                         break;

    //                            // Collectively fetch per-Gaussian data from global to shared
    //                     const int progress = i * render_block_size + thread_rank;
    //                     if (render_g_range.x + progress < render_g_range.y) {
    //                         unsigned coll_id = b_point_list(render_g_range.x + progress);
    //                         assert(coll_id < n_gaussians);
    //                         collected_id[thread_rank] = coll_id;
    //                         collected_centroid[thread_rank] = data.gm_centroids(coll_id);
    //                         collected_inv_cov3[thread_rank] = g_inverse_filtered_cov3d(coll_id);
    //                         collected_3d_masses[thread_rank] = g_filtered_masses(coll_id);
    //                     }
    //                     __syncthreads();

    //                     if (done)
    //                         continue;

    //                            // Iterate over current batch
    //                     for (unsigned j = 0; j < min(render_block_size, n_toDo); j++) {
    //                         const auto inv_cov = collected_inv_cov3[j];
    //                         const auto gaussian1d = gaussian::intersect_with_ray_inv_C(collected_centroid[j], inv_cov, ray);
    //                         const auto centroid = gaussian1d.centre;
    //                         const auto variance = gaussian1d.C + vol_marcher::config::workaround_variance_add_along_ray;
    //                         const auto sd = stroke::sqrt(variance);
    //                         const auto inv_sd = 1 / sd;
    //                         const auto mass_on_ray = gaussian1d.weight * collected_3d_masses[j];

    //                         if (stroke::isnan(gaussian1d.centre))
    //                             continue;
    //                         if (mass_on_ray < 1.1f / 255.f || mass_on_ray > 1'000)
    //                             continue;
    //                         if (variance <= 0 || stroke::isnan(variance) || stroke::isnan(mass_on_ray) || mass_on_ray > 100'000)
    //                             continue; // todo: shouldn't happen any more after implementing AA?

    //                         const auto mass_in_bins = mass_on_ray * gaussian::integrate_normalised_inv_SD(centroid, inv_sd, { bin_borders[0], bin_borders[bin_borders.size() - 1] });

    //                         if (mass_in_bins < 0.0001f) { // performance critical
    //                             continue;
    //                         }
    //                         dbg_mass_in_bins_closeed += mass_in_bins;

    //                         auto cdf_start = gaussian::cdf_inv_SD(centroid, inv_sd, current_large_step_start);
    //                         for (auto k = 0u; k < bin_borders.size() - 1; ++k) {
    //                             // const auto left = bin_borders[k];
    //                             const auto right = bin_borders[k + 1];
    //                             // const auto position = (left + right) / 2;
    //                             // const auto delta_t = (right - left);
    //                             const auto cdf_end = gaussian::cdf_inv_SD(centroid, inv_sd, right);
    //                             const auto mass = stroke::max(0.f, (cdf_end - cdf_start) * mass_on_ray);
    //                             cdf_start = cdf_end;

    //                                    // const auto eval = weight * gaussian::eval_exponential(centroid, variance, position);
    //                                    // const auto mass = stroke::max(0.f, eval * delta_t);
    //                             if (mass < 0.00001f)
    //                                 continue;

    //                             dbg_mass_in_bins_numerik_1 += mass;
    //                             bin_eval[k] += glm::vec4(g_rgb(collected_id[j]) * mass, mass);
    //                         }
    //                     }
    //                 }

    //                 switch (data.debug_render_mode) {
    //                 case vol_marcher::ForwardData::RenderMode::Full: {
    //                     // quadrature rule for bins
    //                     for (auto k = 0u; k < bin_eval.size(); ++k) {
    //                         const auto eval_t = bin_eval[k];
    //                         current_colour += glm::vec<3, float>(eval_t) * current_transparency;
    //                         // current_transparency *= stroke::max(float(0), 1 - eval_t.w);
    //                         current_transparency *= stroke::exp(-eval_t.w);
    //                         // current_mass += eval_t.w * delta_t;
    //                         // current_transparency = stroke::exp(-current_mass);
    //                     }
    //                     break;
    //                 }
    //                 case vol_marcher::ForwardData::RenderMode::Bins: {
    //                     const auto bin = stroke::min(unsigned(data.debug_render_bin), sample_sections.size() - 1);
    //                     const auto mass = sum(bin_eval[bin]);
    //                     // const auto mass = (bin == 0) ? dbg_mass_in_bins_closeed : dbg_mass_in_bins_numerik_1;
    //                     current_colour = glm::vec3(mass * data.max_depth);
    //                     if (mass == 0)
    //                         current_colour = glm::vec3(0, 1.0, 0);
    //                     if (stroke::isnan(mass))
    //                         current_colour = glm::vec3(1, 0, 0.5);
    //                     if (mass < 0)
    //                         current_colour = glm::vec3(1, 0.0, 0);
    //                     current_transparency = 0;
    //                     break;
    //                 }
    //                 case vol_marcher::ForwardData::RenderMode::Depth: {
    //                     const auto bin = stroke::min(unsigned(data.debug_render_bin), sample_sections.size() - 1);
    //                     const auto distance = sample_sections[bin].end;
    //                     // const auto bin = stroke::min(unsigned(data.debug_render_bin), current_large_steps.size() - 1);
    //                     // const auto distance = current_large_steps.data()[bin];
    //                     current_colour = glm::vec3(distance / data.max_depth);
    //                     if (distance == 0)
    //                         current_colour = glm::vec3(0, 1.0, 0);
    //                     if (stroke::isnan(distance))
    //                         current_colour = glm::vec3(1, 0, 0.5);
    //                     if (distance < 0)
    //                         current_colour = glm::vec3(1, 0.5, 0);
    //                     current_transparency = 0;
    //                     break;
    //                 }
    //                 }

    //                 done = done || sample_sections.size() == 0 || current_transparency < 0.001f;
    //                 const int num_done = __syncthreads_count(done);
    //                 if (num_done == render_block_size)
    //                     break;
    //                 // large_stepping_ongoing = false || (current_large_steps.size() == config::n_large_steps && current_transparency > 0.001f);
    //                 current_large_step_start = bin_borders[bin_borders.size() - 1];
    //                 // if (!done)
    //                 // current_large_step_start = sample_sections[sample_sections.size() - 1].end;
    //             }

    //             if (!inside)
    //                 return;
    //             // All threads that treat valid pixel write out their final
    //             const auto final_colour = current_colour + current_transparency * data.background;
    //             data.framebuffer(0, pix.y, pix.x) = final_colour.x;
    //             data.framebuffer(1, pix.y, pix.x) = final_colour.y;
    //             data.framebuffer(2, pix.y, pix.x) = final_colour.z;
    //         });
    // }

    // // preprocess backward, run per Gaussian
    // {
    //     math::Camera<float> camera {
    //         data.view_matrix, data.proj_matrix, focal_x, focal_y, data.tan_fovx, data.tan_fovy, fb_width, fb_height
    //     };

    //     const dim3 block_dim = { 128 };
    //     const dim3 grid_dim = whack::grid_dim_from_total_size({ data.gm_weights.size<0>() }, block_dim);
    //     whack::start_parallel(
    //         whack::Location::Device, grid_dim, block_dim, WHACK_KERNEL(=) {
    //             WHACK_UNUSED(whack_gridDim);
    //             const auto idx = whack_blockIdx.x * whack_blockDim.x + whack_threadIdx.x;
    //             if (idx >= n_gaussians)
    //                 return;

    //             // Initialize touched tiles to 0. If this isn't changed,
    //             // this Gaussian will not be processed further.
    //             g_tiles_touched(idx) = 0;

    //             const auto centroid = data.gm_centroids(idx);
    //             if ((data.view_matrix * glm::vec4(centroid, 1.f)).z < 0.2) // adam doesn't understand, why projection matrix > 0 isn't enough.
    //                 return;

    //             const auto weights = data.gm_weights(idx);
    //             const auto scales = data.gm_cov_scales(idx) * data.cov_scale_multiplier;
    //             const auto rotations = data.gm_cov_rotations(idx);

    //             const auto screen_space_gaussian = math::splat<vol_marcher::config::gaussian_mixture_formulation>(weights, centroid, scales, rotations, camera, 0.3f);

    //             const auto cov3d = math::compute_cov(clamp_cov_scales(data.gm_cov_scales(idx)), data.gm_cov_rotations(idx));

    //             // low pass filter to combat aliasing
    //             const auto filter_kernel_size = glm::distance(centroid, data.cam_poition) * aa_distance_multiplier;
    //             const auto filtered_cov_3d = cov3d + stroke::Cov3_f(filter_kernel_size * filter_kernel_size);
    //             const auto mass = math::weight_to_mass<vol_marcher::config::gaussian_mixture_formulation>(weights, scales + glm::vec3(filter_kernel_size * filter_kernel_size));
    //             if (mass <= 0)
    //                 return; // clipped

    //             // using the more aggressive computation for calculating overlapping tiles:
    //             {
    //                 const glm::uvec2 my_rect = { (int)ceil(3.f * sqrt(screen_space_gaussian.cov[0])), (int)ceil(3.f * sqrt(screen_space_gaussian.cov[2])) };
    //                 g_rects(idx) = my_rect;
    //                 glm::uvec2 rect_min, rect_max;
    //                 getRect(screen_space_gaussian.centroid, my_rect, &rect_min, &rect_max, render_grid_dim);

    //                 const auto tiles_touched = (rect_max.x - rect_min.x) * (rect_max.y - rect_min.y);
    //                 if (tiles_touched == 0)
    //                     return; // clipped
    //                 g_tiles_touched(idx) = tiles_touched;
    //                 g_points_xy_image(idx) = screen_space_gaussian.centroid;
    //             }

    //             const auto inverse_filtered_cov = stroke::inverse(filtered_cov_3d);

    //             // g_depths(idx) = glm::length(data.cam_poition - centroid);
    //             g_depths(idx) = (glm::length(data.cam_poition - centroid) - math::max(scales) * config::gaussian_relevance_sigma / 2);

    //             // convert spherical harmonics coefficients to RGB color.
    //             g_rgb(idx) = computeColorFromSH(data.sh_degree, centroid, data.cam_poition, data.gm_sh_params(idx), &g_rgb_sh_clamped(idx));
    //             g_inverse_filtered_cov3d(idx) = inverse_filtered_cov;
    //             g_filtered_masses(idx) = mass;
    //         });
    // }
    return {};
}
